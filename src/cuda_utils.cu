#include "hip/hip_runtime.h"
/*!
 * libSGM_LiDAR
 * https://github.com/yshry/libSGM_LiDAR
 *
 * Copyright 2025 yshr
 * Released under the MIT license
 * https://github.com/yshry/libSGM_LiDAR/blob/main/LICENSE
 *
 */


#include "internal.h"

#include <hip/hip_runtime.h>

#include "host_utility.h"

namespace
{

__global__ void cast_16bit_8bit_array_kernel(const uint16_t* arr16bits, uint8_t* arr8bits, int num_elements)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < num_elements)
		arr8bits[i] = static_cast<uint8_t>(arr16bits[i]);
}

__global__ void cast_8bit_16bit_array_kernel(const uint8_t* arr8bits, uint16_t* arr16bits, int num_elements)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < num_elements)
		arr16bits[i] = static_cast<uint16_t>(arr8bits[i]);
}

__global__ void shift_right_8bit_for_16bit_array_kernel(const uint16_t* arr_in, uint16_t* arr_out, int num_elements)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < num_elements)
		arr_out[i] = (arr_in[i] >> 8);
}

} // namespace

namespace sgm
{
namespace details
{

void cast_16bit_to_8bit(const DeviceImage& src, DeviceImage& dst)
{
	const int w = src.cols;
	const int h = src.rows;
	dst.create(h, w, SGM_8U, src.step);

	const int num_elements = h * src.step;
	const int block = 1024;
	const int grid = divUp(num_elements, block);

	cast_16bit_8bit_array_kernel<<<grid, block>>>(src.ptr<uint16_t>(), dst.ptr<uint8_t>(), num_elements);
	CUDA_CHECK(hipGetLastError());
}

void cast_8bit_to_16bit(const DeviceImage& src, DeviceImage& dst)
{
	const int w = src.cols;
	const int h = src.rows;
	dst.create(h, w, SGM_16U, src.step);

	const int num_elements = h * src.step;
	const int block = 1024;
	const int grid = divUp(num_elements, block);

	cast_8bit_16bit_array_kernel<<<grid, block>>>(src.ptr<uint8_t>(), dst.ptr<uint16_t>(), num_elements);
	CUDA_CHECK(hipGetLastError());
}

void shift_right_8bit_for_16bit(const DeviceImage& src, DeviceImage& dst)
{
	const int w = src.cols;
	const int h = src.rows;
	dst.create(h, w, SGM_16U, src.step);

	const int num_elements = h * src.step;
	const int block = 1024;
	const int grid = divUp(num_elements, block);

	shift_right_8bit_for_16bit_array_kernel<<<grid, block>>>(src.ptr<uint16_t>(), dst.ptr<uint16_t>(), num_elements);
	CUDA_CHECK(hipGetLastError());
}

} // namespace details
} // namespace sgm
